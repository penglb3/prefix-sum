#include "hip/hip_runtime.h"
#include "common.h"
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <vector>

/* ***********************************
 * Section 1: CUDA Utilities
 ************************************* */

static void handle_error(hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    fprintf(stderr, "%s in %s at line %d\n", hipGetErrorString(err), file,
            line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR(err) (handle_error(err, __FILE__, __LINE__))

inline float cuda_event_tick(const hipEvent_t &start, hipEvent_t &stop) {
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  HANDLE_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
  hipEventRecord(start);
  return milliseconds;
}

enum { DEV_CUDA, DEV_CPU };

class EventTimer {
public:
  hipEvent_t start_cuda, stop_cuda;
  time_point_t time_chrono;
  int event_device;
  explicit EventTimer(int device = DEV_CPU) : event_device(device) {
    HANDLE_ERROR(hipEventCreate(&start_cuda));
    HANDLE_ERROR(hipEventCreate(&stop_cuda));
  }
  float tick(int next_event_device = DEV_CPU) {
    float duration = event_device == DEV_CPU
                         ? chrono_event_tick(time_chrono)
                         : cuda_event_tick(start_cuda, stop_cuda);
    if (event_device == DEV_CPU && next_event_device == DEV_CUDA) {
      hipEventRecord(start_cuda);
    }
    if (event_device == DEV_CUDA && next_event_device == DEV_CPU) {
      time_chrono = std::chrono::high_resolution_clock::now();
    }
    event_device = next_event_device;
    return duration;
  }
};

/* ***********************************
 * Section 2: Kernel functions
 ************************************* */

inline __device__ int min_d(const int a, const int b) {
  return a < b ? a : b;
}

__global__ void scan_block(int *A, const int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    int tmp;
    const int max_s = min_d(blockDim.x, n - blockDim.x * blockIdx.x);
    for (int s = 1; s < max_s; s <<= 1) {
      tmp = A[i - s];
      __syncthreads();
      if (threadIdx.x >= s) {
        A[i] += tmp;
      }
      __syncthreads();
    }
  }
}

__global__ void sum_block_efficient(int *A, const int n) {
  int i = (blockDim.x * blockIdx.x + threadIdx.x) * 2 + 1;
  if (i < n) {
    int s;
    for (s = 1; s < min_d(blockDim.x, n - blockDim.x * blockIdx.x); s <<= 1) {
      if ((threadIdx.x + 1) % s == 0) {
        A[i] += A[i - s];
      }
      __syncthreads();
    }
    if (threadIdx.x == blockDim.x - 1) {
      A[i] += A[i - s];
    }
    for (s >>= 1; s >= 1; s >>= 1) {
      if ((threadIdx.x + 1) % s == 0 && threadIdx.x + s < blockDim.x) {
        A[i + s] += A[i];
      }
      __syncthreads();
    }
  }
}

__global__ void gather_offsets(const int *A, const uint32_t n, int *offsets,
                               const uint64_t sec_size) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if ((i + 1) * sec_size <= n) {
    offsets[i] = A[(i + 1) * sec_size - 1];
  }
}

__global__ void add_offsets(int *A, const uint32_t n, const int *offsets,
                            const uint64_t sec_size, const uint32_t p) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int target_block = blockIdx.x + sec_size / blockDim.x;
  if (i + sec_size < n &&
      target_block / (sec_size * p) == blockIdx.x / (sec_size * p)) {
    A[i + sec_size] += offsets[i / sec_size];
  }
}

/* ***********************************
 * Section 3: CUDA Wrappers
 ************************************* */

inline uint64_t div_up(const uint64_t n, const uint64_t block_size) {
  return (n + block_size - 1) / block_size;
}

const sum_func_t ALGOS[] = {scan_block, sum_block_efficient};
const int UNIT_BLOCK_P[] = {1, 2};

int cuda_wrapper(const int *arr, int *result, const int n, uint8_t type) {
  const int block_size = 1024;
  const int p = UNIT_BLOCK_P[type];
  auto sum_block = ALGOS[type];
  int unit_size = block_size * p;
  int *arr_d, *offsets_d;
  uint64_t n_bytes = n * sizeof(int);
  uint32_t n_block = div_up(n, unit_size);
  std::vector<float> times;
  EventTimer timer;
  timer.tick(); // #0: Alloc memory on memory
  HANDLE_ERROR(hipMalloc((void **)&arr_d, n_bytes));
  HANDLE_ERROR(hipMalloc((void **)&offsets_d, n_block * sizeof(int)));
  times.push_back(timer.tick()); // #1: Copy array to device
  if (offsets_d == NULL || arr_d == NULL) {
    return -1;
  }
  HANDLE_ERROR(hipMemcpy(arr_d, arr, n_bytes, hipMemcpyHostToDevice));
  times.push_back(timer.tick(DEV_CUDA)); // #2: First scan
  // Sum up within each unit
  sum_block<<<n_block, block_size>>>(arr_d, n);
  // A section consists of <unit_size> units
  uint64_t sec_size = unit_size;
  unsigned int n_sec = div_up(n, sec_size * block_size);
  times.push_back(timer.tick(DEV_CUDA)); // #3: section sums
  while (sec_size < n) { // Loop until the whole array become a single section
    // Gather offsets of each unit into offsets_d
    gather_offsets<<<n_sec, block_size>>>(arr_d, n, offsets_d, sec_size);
    // sum up offsets within a section
    sum_block<<<n_sec, block_size>>>(offsets_d, div_up(n, sec_size));
    // For each unit, add corresponding offset. Then every section is done.
    n_block = div_up(n - sec_size, block_size);
    add_offsets<<<n_block, block_size>>>(arr_d, n, offsets_d, sec_size, p);
    // An old section becomes a unit in a new section.
    sec_size *= unit_size;
    n_sec = div_up(n, sec_size * block_size);
  }
  times.push_back(timer.tick()); // #4: Copy results back
  HANDLE_ERROR(hipMemcpy(result, arr_d, n_bytes, hipMemcpyDeviceToHost));
  times.push_back(timer.tick());
  HANDLE_ERROR(hipFree(arr_d));
  HANDLE_ERROR(hipFree(offsets_d));
  for (int i = 0; i < times.size(); i++) {
    printf("#%d: %.3f(ms)\n", i, times[i]);
  }
  printf("[C] %.3f ms\n", times[2] + times[3]);
  return 0;
}